#include "hip/hip_runtime.h"
#include "cir/gpuprocessing/detect_color.cuh"
#include "cir/common/cuda_host_util.cuh"
#include <iostream>

// function is applicable only for HSV model
#define channels 3

namespace cir { namespace gpuprocessing {

void detect_color(uchar* src, const int hueNumber, const int* minHues, const int* maxHues,
		const int minSat, const int maxSat, const int minValue, const int maxValue,
		const int width, const int height, const int step, uchar* dst) {
	int size = hueNumber * sizeof(int);
	int* d_minHues;
	int* d_maxHues;

	HANDLE_CUDA_ERROR(hipMalloc((void**)&d_minHues, size));
	HANDLE_CUDA_ERROR(hipMalloc((void**)&d_maxHues, size));

	HANDLE_CUDA_ERROR(hipMemcpy(d_minHues, minHues, size, hipMemcpyHostToDevice));
	HANDLE_CUDA_ERROR(hipMemcpy(d_maxHues, maxHues, size, hipMemcpyHostToDevice));

	// TODO kernel dims
	dim3 block((width+15)/16, (height+15)/16);
	dim3 thread(16, 16);

	hipEvent_t start;
	hipEvent_t stop;
	HANDLE_CUDA_ERROR(hipEventCreate(&start));
	HANDLE_CUDA_ERROR(hipEventCreate(&stop));
	HANDLE_CUDA_ERROR(hipEventRecord(start, 0)); // TODO stream

	k_detect_color<<<block, thread>>>(src, hueNumber, d_minHues, d_maxHues, minSat, maxSat,
			minValue, maxValue, width, height, step, dst);
	HANDLE_CUDA_ERROR(hipGetLastError());

	HANDLE_CUDA_ERROR(hipEventRecord(stop, 0));
	HANDLE_CUDA_ERROR(hipEventSynchronize(stop));

	float time;
	HANDLE_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
	HANDLE_CUDA_ERROR(hipEventDestroy(start));
	HANDLE_CUDA_ERROR(hipEventDestroy(stop));

	std::cout << "time: " << time << std::endl;

	HANDLE_CUDA_ERROR(hipFree(d_minHues));
	HANDLE_CUDA_ERROR(hipFree(d_maxHues));
}

__global__
void k_detect_color(uchar* src, const int hueNumber, const int* minHues, const int* maxHues,
		const int minSat, const int maxSat, const int minValue, const int maxValue,
		const int width, const int height, const int step, uchar* dst) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x > width || y > height)
		return;

	int pos = x * channels + y * step;

	int hue = dst[pos];
	int sat = dst[pos+1];
	int value = dst[pos+2];

	bool clear = true;

	if(sat >= minSat && sat <= maxSat
			&& value >= minValue && value <= maxValue) {

		for(int i = 0; i < hueNumber; i++) {
			int minHue = minHues[i];
			int maxHue = maxHues[i];

			if(minHue <= maxHue) {
				if(hue >= minHue && hue <= maxHue) {
					clear = false;
					break;
				}
			} else {
				if(hue >= minHue || hue <= maxHue) {
					clear = false;
					break;
				}
			}
		}

	}

	if(clear) {
		dst[pos] = 0;
		dst[pos+1] = 0;
		dst[pos+2] = 0;
	}
}

}}
