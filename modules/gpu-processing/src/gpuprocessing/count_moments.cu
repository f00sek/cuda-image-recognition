#include "hip/hip_runtime.h"
#include <iostream>
#include "cir/common/config.h"
#include "cir/common/cuda_host_util.cuh"
#include "cir/gpuprocessing/count_moments.cuh"
#include "cir/common/logger/Logger.h"

#define RAW_MOMENTS 10
#define IN_PASS 2
#define THREADS_IN_BLOCK 16
#define THREADS_PER_BLOCK THREADS_IN_BLOCK * THREADS_IN_BLOCK

using namespace cir::common;
using namespace cir::common::logger;

namespace cir { namespace gpuprocessing {

void count_raw_moments(uchar* data, int width, int height, int step, double* rawMoments,
		hipStream_t stream) {
	int horizontalBlocks = (width + THREADS_IN_BLOCK - 1) / THREADS_IN_BLOCK;
	int verticalBlocks = (height + THREADS_IN_BLOCK - 1) / THREADS_IN_BLOCK;
	int totalBlocks = horizontalBlocks * verticalBlocks;
	int totalToAlloc = totalBlocks * IN_PASS;

	long* blockSums;
	hipHostAlloc((void**) &blockSums, sizeof(long) * totalToAlloc, hipHostMallocDefault);

	long* d_blockSums;
	HANDLE_CUDA_ERROR(hipMalloc((void**) &d_blockSums, sizeof(long) * totalToAlloc));

	for(int i = 0; i < RAW_MOMENTS; i++) {
		rawMoments[i] = 0.;
	}

	// TODO kernel dims
	dim3 blocks(horizontalBlocks, verticalBlocks);
	dim3 threads(THREADS_IN_BLOCK, THREADS_IN_BLOCK);

	for(int pass = 0; pass < (RAW_MOMENTS + IN_PASS - 1) / IN_PASS; pass++) {
		for(int i = 0; i < totalToAlloc; i++) {
			blockSums[i] = 0;
		}

		HANDLE_CUDA_ERROR(hipMemcpyAsync(d_blockSums, blockSums, sizeof(long) * totalToAlloc, hipMemcpyHostToDevice,
				stream));

		KERNEL_MEASURE_START(stream)

		k_count_raw_moment<<<blocks, threads, 0, stream>>>(data, width, height, step, pass, d_blockSums);
		HANDLE_CUDA_ERROR(hipGetLastError());

		KERNEL_MEASURE_END("Count Hu moments", stream)

		HANDLE_CUDA_ERROR(hipMemcpyAsync(blockSums, d_blockSums, sizeof(long) * totalToAlloc, hipMemcpyDeviceToHost,
				stream));

		HANDLE_CUDA_ERROR(hipStreamSynchronize(stream));

		for(int j = 0; j < totalBlocks; j++) {
			for(int i = 0; i < IN_PASS; i++) {
				rawMoments[pass * IN_PASS + i] += blockSums[j * IN_PASS + i];
			}
		}
	}

	HANDLE_CUDA_ERROR(hipFree(d_blockSums));
	HANDLE_CUDA_ERROR(hipHostFree(blockSums));
}

__global__
void k_count_raw_moment(uchar* data, int width, int height, int step, int pass, long* blockSums) {
	__shared__ long cache[THREADS_PER_BLOCK * IN_PASS];
	int tid = threadIdx.x + blockDim.x * threadIdx.y;
	int cacheIdx = tid * IN_PASS;
	for(int i = 0; i < IN_PASS; i++) {
		cache[cacheIdx + i] = 0;
	}

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x >= width)
		return;

	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(y >= height)
		return;

	int idx = x + y * step;
#if MOMENTS_BINARY
	int pixel = data[idx] == 0 ? 0 : 1;
#else
	int pixel = data[idx];
#endif

	if(pass == 0) {
	/* M00 */ cache[cacheIdx + 0] = pixel;
	/* M01 */ cache[cacheIdx + 1] = pixel * y;
	} else if(pass == 1) {
	/* M10 */ cache[cacheIdx + 0] = pixel * x;
	/* M11 */ cache[cacheIdx + 1] = pixel * x * y;
	} else if(pass == 2) {
	/* M02 */ cache[cacheIdx + 0] = pixel * y * y;
	/* M20 */ cache[cacheIdx + 1] = pixel * x * x;
	} else if(pass == 3) {
	/* M21 */ cache[cacheIdx + 0] = pixel * x * x * y;
	/* M12 */ cache[cacheIdx + 1] = pixel * x * y * y;
	} else if(pass == 4) {
	/* M30 */ cache[cacheIdx + 0] = pixel * x * x * x;
	/* M03 */ cache[cacheIdx + 1] = pixel * y * y * y;
	}

	__syncthreads();

	for(int j = THREADS_PER_BLOCK / 2; j != 0; j /= 2) {
		if(tid < j) {
			for(int i = 0; i < IN_PASS; i++) {
				cache[cacheIdx + i] += cache[cacheIdx + i + j * IN_PASS];
			}
		}
		__syncthreads();
	}

	if(tid == 0) {
		for(int i = 0; i < IN_PASS; i++) {
			blockSums[i + (blockIdx.x + blockIdx.y * gridDim.x) * IN_PASS] = cache[i];
		}
	}
}

}}
