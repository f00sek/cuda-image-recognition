#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <iostream>
#include <iomanip>
#include <cstdio>
#include "cir/gpuprocessing/region_splitting_segmentate.cuh"
#include "cir/common/cuda_host_util.cuh"

#define CHANNELS 3

using namespace cir::common;
using namespace cir::common::logger;

namespace cir { namespace gpuprocessing {

element* elements;
element* d_elements;

Segment* segments;
Segment* d_segments;

void region_splitting_segmentate_init(int width, int height) {
	elements = (element*) malloc(sizeof(element) * width * height);
	segments = (Segment*) malloc(sizeof(Segment) * width * height);

	HANDLE_CUDA_ERROR(hipMalloc((void**) &d_elements, sizeof(element) * width * height));
	HANDLE_CUDA_ERROR(hipMalloc((void**) &d_segments, sizeof(Segment) * width * height));
}

bool is_segment_applicable(Segment* segment) {
	return true;
}

SegmentArray* region_splitting_segmentate(uchar* data, int step, int channels, int width, int height) {
	for(int i = 0; i < width * height; i++) {
		elements[i].id = i;
		elements[i].valid = true;

		segments[i] = createSimpleSegment(i % width, i / width);
	}

	HANDLE_CUDA_ERROR(hipMemcpy(d_elements, elements, sizeof(element) * width * height, hipMemcpyHostToDevice));
	HANDLE_CUDA_ERROR(hipMemcpy(d_segments, segments, sizeof(Segment) * width * height, hipMemcpyHostToDevice));

	int THREADS = 16;

	dim3 blocks((width+THREADS-1)/THREADS, (height+THREADS-1)/THREADS);
	dim3 threads(THREADS, THREADS);

	k_remove_empty_segments<<<blocks, threads>>>(data, width, height, step, d_elements);

	int greaterDim = width > height ? width : height;

	HANDLE_CUDA_ERROR(hipGetLastError());

	for(int i = 1; i < greaterDim; i = 2 * i) {
		int block_width = i;
		int block_height = i;

		dim3 blocks((width+i*THREADS-1)/(i*THREADS), (height+i*THREADS-1)/(i*THREADS));
		dim3 threads(THREADS, THREADS);
		KERNEL_MEASURE_START

		k_region_splitting_segmentate<<<blocks, threads>>>(data, d_elements, d_segments, step,
				channels, width, height, block_width, block_height);
		HANDLE_CUDA_ERROR(hipGetLastError());

		KERNEL_MEASURE_END("Segmentate")

//		HANDLE_CUDA_ERROR(hipMemcpy(elements, d_elements, sizeof(element) * width * height, hipMemcpyDeviceToHost));
//		for(int x = 0; x < width; x++) {
//			for(int y = 0; y < height; y++) {
//				std::cout << std::setw(6) << elements[x*height + y].id << " ";
//			}
//
//			std::cout << std::endl;
//		}
//		std::cout << "-----------" << std::endl;
//		HANDLE_CUDA_ERROR(hipDeviceSynchronize());
	}

	HANDLE_CUDA_ERROR(hipMemcpy(elements, d_elements, sizeof(element) * width * height, hipMemcpyDeviceToHost));
	HANDLE_CUDA_ERROR(hipMemcpy(segments, d_segments, sizeof(Segment) * width * height, hipMemcpyDeviceToHost));

	int foundSegmentsSize = 0;
	for(int j = 0; j < width*height; j++) {
		if(elements[j].valid && is_segment_applicable(&(segments[j]))) {
			foundSegmentsSize++;
		}
	}

	SegmentArray* segmentArray = (SegmentArray*) malloc(sizeof(SegmentArray));
	segmentArray->size = foundSegmentsSize;

	if(foundSegmentsSize > 0) {
		Segment** appliedSegments = (Segment**) malloc(sizeof(Segment*) * foundSegmentsSize);
		int currentSegmentIndex = 0;
		for(int j = 0; j < width*height; j++) {
			if(elements[j].valid && is_segment_applicable(&(segments[j]))) {
				Segment segment = segments[j];
				appliedSegments[currentSegmentIndex++] = copySegment(&segment);
			}
		}
		segmentArray->segments = appliedSegments;
	} else {
		segmentArray->segments = NULL;
	}

	return segmentArray;
}

void region_splitting_segmentate_shutdown() {
	HANDLE_CUDA_ERROR(hipFree(d_elements));
	HANDLE_CUDA_ERROR(hipFree(d_segments));

	free(elements);
	free(segments);
}

__global__
void k_remove_empty_segments(uchar* data, int width, int height, int step, element* elements) {
	int ai_x = blockDim.x * blockIdx.x + threadIdx.x;
	if(ai_x >= width)
		return;

	int ai_y = blockDim.y * blockIdx.y + threadIdx.y;
	if(ai_y >= height)
		return;

	int di = ai_x * CHANNELS + ai_y * step;
	uchar saturation = data[di+1];
	uchar value = data[di+2];

	if(saturation == 0 && value == 0) {
		element* elem = &(elements[ai_x + width * ai_y]);
		elem->id = -1;
		elem->valid = false;
	}
}

// tlb - top left block
// brb - bottom right block
// di_ - data index
// ai - array index
__global__
void k_region_splitting_segmentate(uchar* data, element* elements, Segment* segments,
		int step, int channels, int width, int height, int block_width, int block_height) {
	int ai_x = blockDim.x * blockIdx.x + threadIdx.x;
	int ai_y = blockDim.y * blockIdx.y + threadIdx.y;

	if(ai_x % 2 != 0 || ai_y % 2 != 0)
		return;

	ai_x = ai_x * block_width;
	ai_y = ai_y * block_height;

	if(ai_x >= width || ai_y >= height)
		return;

	// top left and top right
	int di_tlb_top_right_x = (ai_x + block_width - 1) * channels + ai_y * step;
	int ai_lb_top_right_x = ai_x + block_width - 1;

	d_merge_blocks_horizontally(di_tlb_top_right_x, step, channels, ai_lb_top_right_x, width, height,
			ai_y, data, elements, segments, block_width, block_height);

	// bottom left and bottom right
	int di_blb_top_right_x = di_tlb_top_right_x + block_height * step;
	int blb_ai_y = ai_y + block_height;

	d_merge_blocks_horizontally(di_blb_top_right_x, step, channels, ai_lb_top_right_x, width, height,
			blb_ai_y, data, elements, segments, block_width, block_height);

	// top left/right and bottom left/right
	int di_tb_bottom_left_y = ai_x * channels + (ai_y + block_height - 1) * step;
	d_merge_blocks_vertically(di_tb_bottom_left_y, step, channels, ai_x, width, height, ai_y + block_height - 1,
			data, elements, segments, block_width, block_height);
}

__device__
void d_merge_blocks_horizontally(int di_lb_top_right_x, int step,
		int channels, int ai_x, int width, int height, int ai_y, uchar* data, element* elements,
		Segment* segments, int block_width, int block_height) {

	for (int i = 0; i < block_height; i++) {
		int di_tlb_right = di_lb_top_right_x + i * step;
		int di_trb_left = di_tlb_right + channels;
		int ai_tlb = ai_x + width * (i + ai_y);
		int ai_trb = ai_tlb + 1;

		if(ai_trb % width < ai_tlb % width || ai_trb > width * height)
			return;

		if (!d_is_empty(data, di_tlb_right) && !d_is_empty(data, di_trb_left)) {
			element* left_elem = &(elements[ai_tlb]);
			element* right_elem = &(elements[ai_trb]);

			int left_elem_id = left_elem->id;
			int right_elem_id = right_elem->id;

			for(int j = 0; j < block_height; j++) {
				int ai_tlb_right = ai_x + width * j + ai_y * width;
				d_try_merge(ai_tlb_right, right_elem_id, left_elem_id, width, height,
						elements, segments, false);

				int ai_trb_left = ai_tlb_right + 1;
				d_try_merge(ai_trb_left, right_elem_id, left_elem_id, width, height,
						elements, segments);

				int x_trb_left = ai_trb_left % width;
				int normalized_width = x_trb_left + block_width - 1 > width ? width - x_trb_left + 1 : block_width;

				int ai_trb_right = ai_trb_left + normalized_width - 1;
				d_try_merge(ai_trb_right, right_elem_id, left_elem_id, width, height,
						elements, segments);

				int ai_tlb_left = ai_trb_left - block_width;
				d_try_merge(ai_tlb_left, right_elem_id, left_elem_id, width, height,
						elements, segments, false);
			}

			for(int j = 0; j < block_width; j++) {
				int ai_trb_top = ai_x + j + 1 + ai_y * width;
				d_try_merge(ai_trb_top, right_elem_id, left_elem_id, width, height,
						elements, segments);

				int ai_tlb_top = ai_trb_top - block_width;
				d_try_merge(ai_tlb_top, right_elem_id, left_elem_id, width, height,
						elements, segments, false);

				int y_trb_top = ai_trb_top / width;
				int normalized_height = y_trb_top + block_height - 1 > height ? height - y_trb_top + 1 : block_height;

				int ai_trb_bottom = (normalized_height - 1) * width + ai_trb_top;
				d_try_merge(ai_trb_bottom, right_elem_id, left_elem_id, width, height,
						elements, segments);

				int ai_tlb_bottom = ai_trb_bottom - block_width;
				d_try_merge(ai_tlb_bottom, right_elem_id, left_elem_id, width, height,
						elements, segments, false);
			}
		}
	}
}

__device__
void d_merge_blocks_vertically(int di_lb_bottom_left_y, int step,
		int channels, int ai_x, int width, int height, int ai_y, uchar* data, element* elements,
		Segment* segments, int block_width, int block_height) {

	for (int i = 0; i < 2*block_width; i++) {
		int di_tlb_bottom = di_lb_bottom_left_y + i * channels;
		int di_blb_top = di_tlb_bottom + step;
		int ai_tb = ai_x + i + width * ai_y;
		int ai_bb = ai_tb + width;

		if(ai_bb / width > height || ai_bb > width * height)
			return;

		if (!d_is_empty(data, di_tlb_bottom) && !d_is_empty(data, di_blb_top)) {
			element* top_elem = &(elements[ai_tb]);
			element* bottom_elem = &(elements[ai_bb]);

			int top_elem_id = top_elem->id;
			int bottom_elem_id = bottom_elem->id;

			for(int j = 0; j < 2*block_width; j++) {
				int ai_bb_top = ai_x + width + j + ai_y * width;
				d_try_merge(ai_bb_top, bottom_elem_id, top_elem_id, width, height,
						elements, segments);

				int ai_tb_top = ai_bb_top - block_height * width;
				d_try_merge(ai_tb_top, bottom_elem_id, top_elem_id, width, height,
						elements, segments, false);

				int y_bb_top = ai_bb_top / width;
				int normalized_height = y_bb_top + block_height - 1 > height ? height - y_bb_top + 1 : block_height;

				int ai_bb_bottom = ai_bb_top + (normalized_height-1) * width;
				d_try_merge(ai_bb_bottom, bottom_elem_id, top_elem_id, width, height,
						elements, segments);

				int ai_tb_bottom = ai_bb_bottom - block_height * width;
				d_try_merge(ai_tb_bottom, bottom_elem_id, top_elem_id, width, height,
						elements, segments, false);
			}

			for(int j = 0; j < block_height; j++) {
				int ai_bb_left = ai_x + (j+1) * width + ai_y * width;
				d_try_merge(ai_bb_left, bottom_elem_id, top_elem_id, width, height,
						elements, segments);

				int ai_tb_left = ai_bb_left - block_height * width;
				d_try_merge(ai_tb_left, bottom_elem_id, top_elem_id, width, height,
						elements, segments, false);

				int x_bb_left = ai_bb_left % width;
				int normalized_width = x_bb_left + 2*block_width - 1 > width ? width - x_bb_left + 1 : 2 * block_width;

				int ai_bb_right = ai_bb_left + normalized_width - 1;
				d_try_merge(ai_bb_right, bottom_elem_id, top_elem_id, width, height,
						elements, segments);

				int ai_tb_right = ai_bb_right - block_height * width;
				d_try_merge(ai_tb_right, bottom_elem_id, top_elem_id, width, height,
						elements, segments, false);
			}
		}
	}
}

__device__
void d_try_merge(int idx, int current_elem_id, int id_to_set, int width, int height,
		element* elements, Segment* segments, bool invalidate_all) {
	if(idx < width * height) {
		element* elem = &(elements[idx]);
		if(elem->id == current_elem_id) {
			Segment* segm1 = &(segments[id_to_set]);
			Segment* segm2 = &(segments[elem->id]);
			if(id_to_set != elem->id) {
				element* elemToInvalidate = &(elements[elem->id]);
				if(invalidate_all)
					elemToInvalidate->valid = false;
				elemToInvalidate->id = id_to_set;
			}
			d_merge_segments(segm1, segm2);
			if(invalidate_all)
				elem->valid = false;
			elem->id = id_to_set;
		}
	}
}

__device__
bool d_is_empty(uchar* data, int addr) {
	return data[addr+1] == 0 && data[addr+2] == 0;
}

__device__
void d_merge_segments(Segment* segm1, Segment* segm2) {
	if(segm1->leftX < segm2->leftX) {
		segm2->leftX = segm1->leftX;
	} else {
		segm1->leftX = segm2->leftX;
	}

	if(segm1->rightX > segm2->rightX) {
		segm2->rightX = segm1->rightX;
	} else {
		segm1->rightX = segm2->rightX;
	}

	if(segm1->bottomY > segm2->bottomY) {
		segm2->bottomY = segm1->bottomY;
	} else {
		segm1->bottomY = segm2->bottomY;
	}

	if(segm1->topY < segm2->topY) {
		segm2->topY = segm1->topY;
	} else {
		segm1->topY = segm2->topY;
	}
}

/*
// tlb - top left block
// brb - bottom right block
// di_ - data index
// ai - array index
__global__
void k_region_splitting_segmentate(uchar* data, element* elements, Segment* segments, int step,
		int channels, int width, int height, int block_width, int block_height) {
	int ai_x = blockIdx.x * blockDim.x + threadIdx.x;
	if(ai_x % 2 != 0)
		return;

	int ai_y = blockIdx.y * blockDim.y + threadIdx.y;
	if(ai_y % 2 != 0)
		return;

	ai_x = ai_x * block_width;
	if(ai_x >= width)
		return;

	ai_y = ai_y * block_height;
	if(ai_y >= height)
		return;

	int merged_y_start_idx = ai_x + ai_y * width;
	int merged_y_current_idx = merged_y_start_idx;

	int merged_x_start_idx = ai_x + ai_y * width;
	int merged_x_current_idx = merged_x_start_idx;

	// top left and top right
	int di_tlb_top_right_x = (ai_x + block_width - 1) * channels + ai_y * step;
	int ai_lb_top_right_x = ai_x + block_width - 1;

	d_merge_blocks_horizontally(di_tlb_top_right_x, step, channels, ai_lb_top_right_x, width, height,
			ai_y, merged_y_start_idx, &merged_y_current_idx, data, elements,
			merged_y, block_height);

	// bottom left and bottom right
	int di_blb_top_right_x = di_tlb_top_right_x + block_height * step;
	int blb_ai_y = ai_y + block_height;

	d_merge_blocks_horizontally(di_blb_top_right_x, step, channels, ai_lb_top_right_x, width, height,
			blb_ai_y, merged_y_start_idx, &merged_y_current_idx, data, elements,
			merged_y, block_height);

	// top left/right and bottom left/right
	int di_tb_bottom_left_y = ai_x * channels + (ai_y + block_height - 1) * step;
	d_merge_blocks_vertically(di_tb_bottom_left_y, step, channels, ai_x, width, height, ai_y + block_height - 1,
			merged_x_start_idx, &merged_x_current_idx, data, elements, merged_x, block_width);
}

__device__
void d_merge_blocks_horizontally(int di_lb_top_right_x, int step, int channels,
		int ai_x, int width, int height, int ai_y, int merged_y_start_idx,
		int *merged_y_current_idx, uchar* data, element* elements,
		elements_pair* merged_y, int block_height) {

	for (int i = 0; i < block_height; i++) {
		int di_tlb_right = di_lb_top_right_x + i * step;
		int di_trb_left = di_tlb_right + channels;
		int ai_tlb = ai_x + width * (i + ai_y);
		int ai_trb = ai_tlb + 1;

		if(ai_trb / height >= width)
			return;

		if (!d_is_empty(data, di_tlb_right) && !d_is_empty(data, di_trb_left)) {
			element* left_elem = &(elements[ai_tlb]);
			element* right_elem = &(elements[ai_trb]);
			if (d_already_merged(merged_y, merged_y_start_idx, *merged_y_current_idx, left_elem, right_elem))
				continue;

			d_merge_elements(elements, left_elem, right_elem, width);
			merged_y[*merged_y_current_idx].id1 = left_elem->id;
			merged_y[*merged_y_current_idx].id2 = right_elem->id;
			*merged_y_current_idx += 1;
		}
	}
}

__device__
void d_merge_blocks_vertically(int di_lb_bottom_left_y, int step, int channels,
		int ai_x, int width, int height, int ai_y, int merged_x_start_idx,
		int *merged_x_current_idx, uchar* data, element* elements,
		elements_pair* merged_x, int block_width) {

	for (int i = 0; i < 2*block_width; i++) {
		int di_tlb_bottom = di_lb_bottom_left_y + i * channels;
		int di_blb_top = di_tlb_bottom + step;
		int ai_tb = ai_x + i + width * ai_y;
		int ai_bb = ai_tb + width;

		if(ai_bb / width >= height)
			return;

		if (!d_is_empty(data, di_tlb_bottom) && !d_is_empty(data, di_blb_top)) {
			element* top_elem = &(elements[ai_tb]);
			element* bottom_elem = &(elements[ai_bb]);
			if (d_already_merged(merged_x, merged_x_start_idx, *merged_x_current_idx, top_elem, bottom_elem))
				continue;

			d_merge_elements(elements, top_elem, bottom_elem, width);
			merged_x[*merged_x_current_idx].id1 = top_elem->id;
			merged_x[*merged_x_current_idx].id2 = bottom_elem->id;
			*merged_x_current_idx += 1;
		}
	}
}

__device__
void d_merge_elements(element* elements, element* e1, element* e2, int width) {
	(&(elements[e1->next]))->prev = e2->prev;
	(&(elements[e2->prev]))->next = e1->next;
	e1->next = width * e2->point.y + e2->point.x;
	e2->prev = width * e1->point.y + e1->point.x;

	e2->id = e1->id;

	// TODO very ineffective
	int end = elements[e2->next].prev; // converts element to its position
	for(int i = e2->next; i != end;) {
		element* elem = &(elements[i]);
		elem->id = e1->id;
		i = elem->next;
	}
}

__device__
bool d_is_empty(uchar* data, int addr) {
	return data[addr+1] == 0 && data[addr+2] == 0; // TODO channels?
}

__device__
bool d_already_merged(elements_pair* merged, int merged_start_idx, int merged_last_idx,
		element* e1, element* e2) {
	for(int i = merged_start_idx; i < merged_last_idx; i++) {
		if(merged[i].id1 == e1->id && merged[i].id2 == e2->id)
			return true;
	}

	return false;
}*/

}}
