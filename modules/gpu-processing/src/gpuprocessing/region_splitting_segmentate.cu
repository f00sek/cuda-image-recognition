#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <iostream>
#include "cir/gpuprocessing/region_splitting_segmentate.cuh"

using namespace cir::common;

namespace cir { namespace gpuprocessing {

void region_splitting_segmentate(uchar* data, int step, int channels, int width, int height) {
	element* elements = (element*)malloc(sizeof(element) * width * height);
	for(int i = 0; i < width*height; i++) {
		elements[i].id = i;
		elements[i].next = i;
		elements[i].prev = i;
		elements[i].point.x = i % width;
		elements[i].point.y = i / width;
		elements[i].v = i;
	}

	elements_pair* merged_y = (elements_pair*) malloc(sizeof(elements_pair) * width * height);
	elements_pair* merged_x = (elements_pair*) malloc(sizeof(elements_pair) * width * height);
	for(int i = 0; i < width*height; i++) {
		merged_x[i].id1 = -1;
		merged_y[i].id2 = -1;
	}

	element* d_elements;
	elements_pair* d_merged_y;
	elements_pair* d_merged_x;

	hipMalloc((void**) &d_elements, sizeof(element) * width * height);
	hipMalloc((void**) &d_merged_y, sizeof(elements_pair) * width * height);
	hipMalloc((void**) &d_merged_x, sizeof(elements_pair) * width * height);

	hipMemcpy(d_elements, elements, sizeof(element) * width * height, hipMemcpyHostToDevice);
	hipMemcpy(d_merged_y, merged_y, sizeof(elements_pair) * width * height, hipMemcpyHostToDevice);
	hipMemcpy(d_merged_x, merged_x, sizeof(elements_pair) * width * height, hipMemcpyHostToDevice);

	dim3 init_blocks(width, height);

	int greaterDim = width > height ? width : height;

	for(int i = 1; i < greaterDim; i = 2 * i) {
		dim3 blocks((width+i-1)/i, (height+i-1)/i);
		dim3 threads(1, 1);
		k_region_splitting_segmentate<<<blocks, threads>>>(data, d_merged_y, d_merged_x, d_elements,
				step, channels, width, height);
		hipMemcpy(elements, d_elements, sizeof(element) * width * height, hipMemcpyDeviceToHost);
		hipMemcpy(merged_x, d_merged_x, sizeof(int) * width * height, hipMemcpyDeviceToHost);
		hipMemcpy(merged_y, d_merged_y, sizeof(int) * width * height, hipMemcpyDeviceToHost);
		for(int x = 0; x < width; x++) {
			for(int y = 0; y < height; y++) {
				std::cerr << elements[x*height + y].id << " ";
			}

			std::cerr << std::endl;
		}
		std::cerr << "-----------" << std::endl;
		hipDeviceSynchronize();
	}

	hipFree(d_elements);
	hipFree(d_merged_y);
	hipFree(d_merged_x);

	free(merged_y);
	free(merged_x);
	free(elements);
}

// tlb - top left block
// brb - bottom right block
// di_ - data index
// ai - array index
__global__
void k_region_splitting_segmentate(uchar* data, elements_pair* merged_y, elements_pair* merged_x,
		element* elements, int step, int channels, int width, int height) {
	int ai_x = blockIdx.x * blockDim.x + threadIdx.x;
	int ai_y = blockIdx.y * blockDim.y + threadIdx.y;

	if(ai_x % 2 != 0 || ai_y % 2 != 0)
		return;

	int block_width = width / gridDim.x;
	int block_height = height / gridDim.y;

	ai_x = ai_x * block_width;
	ai_y = ai_y * block_height;

	int merged_y_start_idx = ai_x + ai_y * blockDim.x * gridDim.x;
	int merged_y_current_idx = merged_y_start_idx;

	int merged_x_start_idx = ai_x + ai_y * blockDim.x * gridDim.x;
	int merged_x_current_idx = merged_x_start_idx;

	// top left and top right
	int di_tlb_top_right_x = (ai_x + block_width - 1) * channels + ai_y * step;
	int ai_lb_top_right_x = ai_x + block_width - 1;

	d_merge_blocks_horizontally(di_tlb_top_right_x, step, channels, ai_lb_top_right_x, width,
			ai_y, merged_y_start_idx, &merged_y_current_idx, data, elements,
			merged_y);

	// bottom left and bottom right
	int di_blb_top_right_x = di_tlb_top_right_x + block_height * step;
	int blb_ai_y = ai_y + block_height;

	d_merge_blocks_horizontally(di_blb_top_right_x, step, channels, ai_lb_top_right_x, width,
			blb_ai_y, merged_y_start_idx, &merged_y_current_idx, data, elements,
			merged_y);

	// top left/right and bottom left/right
	int di_tb_bottom_left_y = ai_x * channels + (ai_y + block_height - 1) * step;
	d_merge_blocks_vertically(di_tb_bottom_left_y, step, channels, ai_x, width, ai_y + block_height - 1,
			merged_x_start_idx, &merged_x_current_idx, data, elements, merged_x);
}

__device__
void d_merge_blocks_horizontally(int di_lb_top_right_x, int step, int channels,
		int ai_x, int width, int ai_y, int merged_y_start_idx,
		int *merged_y_current_idx, uchar* data, element* elements,
		elements_pair* merged_y) {

	int block_height = width / gridDim.x; // TODO

	for (int i = 0; i < block_height; i++) {
		int di_tlb_right = di_lb_top_right_x + i * step;
		int di_trb_left = di_tlb_right + channels;
		int ai_tlb = ai_x + width * (i + ai_y);
		int ai_trb = ai_tlb + 1;
		if (!d_is_empty(data, di_tlb_right) && !d_is_empty(data, di_trb_left)) {
			element* left_elem = &(elements[ai_tlb]);
			element* right_elem = &(elements[ai_trb]);
			if (d_already_merged(merged_y, merged_y_start_idx, *merged_y_current_idx, left_elem, right_elem))
				continue;

			d_merge_elements(elements, left_elem, right_elem, width);
			merged_y[*merged_y_current_idx].id1 = left_elem->id;
			merged_y[*merged_y_current_idx].id2 = right_elem->id;
			*merged_y_current_idx += 1;
		}
	}
}

__device__
void d_merge_blocks_vertically(int di_lb_bottom_left_y, int step, int channels,
		int ai_x, int width, int ai_y, int merged_x_start_idx,
		int *merged_x_current_idx, uchar* data, element* elements,
		elements_pair* merged_x) {

	int block_width = width / gridDim.x; // TODO

	for (int i = 0; i < 2*block_width; i++) {
		int di_tlb_bottom = di_lb_bottom_left_y + i * channels;
		int di_blb_top = di_tlb_bottom + step;
		int ai_tb = ai_x + i + width * ai_y;
		int ai_bb = ai_tb + width;
		if (!d_is_empty(data, di_tlb_bottom) && !d_is_empty(data, di_blb_top)) {
			element* top_elem = &(elements[ai_tb]);
			element* bottom_elem = &(elements[ai_bb]);
			if (d_already_merged(merged_x, merged_x_start_idx, *merged_x_current_idx, top_elem, bottom_elem))
				continue;

			d_merge_elements(elements, top_elem, bottom_elem, width);
			merged_x[*merged_x_current_idx].id1 = top_elem->id;
			merged_x[*merged_x_current_idx].id2 = bottom_elem->id;
			*merged_x_current_idx += 1;
		}
	}
}

__device__
void d_merge_elements(element* elements, element* e1, element* e2, int width) {
	(&(elements[e1->next]))->prev = e2->prev;
	(&(elements[e2->prev]))->next = e1->next;
	e1->next = width * e2->point.y + e2->point.x;
	e2->prev = width * e1->point.y + e1->point.x;

	e2->id = e1->id;

	// TODO very ineffective
	int end = elements[e2->next].prev; // converts element to its position
	for(int i = e2->next; i != end;) {
		element* elem = &(elements[i]);
		elem->id = e1->id;
		i = elem->next;
	}

}

__device__
bool d_is_empty(uchar* data, int addr) {
	return data[addr+1] == 0 && data[addr+2] == 0; // TODO channels?
}

__device__
bool d_already_merged(elements_pair* merged, int merged_start_idx, int merged_last_idx,
		element* e1, element* e2) {
	for(int i = merged_start_idx; i < merged_last_idx; i++) {
		if(merged[i].id1 == e1->id && merged[i].id2 == e2->id)
			return true;
	}

	return false;
}

}}
